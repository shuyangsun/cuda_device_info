#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdio>

#ifdef __HIPCC__

__host__
void CheckCUDAErr(const std::initializer_list<const hipError_t>& errors);
__host__
void PrintGPUDeviceInfo();

#endif

int main(int argc, const char* argv[]) {

#ifdef __HIPCC__
	PrintGPUDeviceInfo();
#else
	fprintf(stderr, "Error: please use nvcc to compile the program.");
#endif
	return 0;
}


#ifdef __HIPCC__

__host__
void PrintGPUDeviceInfo() {
	printf("------------------------------------------------------------------------------\n");
	printf("                               CUDA Device Info                               \n");
	printf("------------------------------------------------------------------------------\n");
	int device_count {};
	CheckCUDAErr({hipGetDeviceCount(&device_count)});

	if (device_count == 0) {
		fprintf(stderr, "No available CUDA device\n");
	} else {
		printf("Detected %u CUDA capable device%s\n", device_count, device_count <= 1 ? "" : "s");
		for (unsigned int device_idx = 0; device_idx < device_count; ++device_idx) {
			printf("\n");
			hipSetDevice(device_idx);

			hipDeviceProp_t device_prop {};
			int driver_version {};
			int runtime_version {};

			CheckCUDAErr({
				hipGetDeviceProperties(&device_prop, device_idx),
				hipDriverGetVersion(&driver_version),
				hipRuntimeGetVersion(&runtime_version)
			});

			printf("Device %u: \"%s\"\n", device_idx, device_prop.name);
			printf("\tCUDA driver / runtime version: %d.%d / %d.%d\n",
					driver_version/1000, (driver_version%100)/10, runtime_version/1000, (runtime_version%100)/10);
			printf("\tCUDA computing capability: %d.%d\n", device_prop.major, device_prop.minor);
			printf("\tTotal amount of global memory: %.2f GB (%lu bytes)\n",
					device_prop.totalGlobalMem/std::pow(1024.0f, 3.0f), device_prop.totalGlobalMem);
			printf("\tGPU clock rate: %.2f GHz (%.0f MHz)\n",
					device_prop.clockRate * 1e-6f, device_prop.clockRate * 1e-3f);
			printf("\tMemory clock rate: %.2f GHz (%.0f MHz) \n",
					device_prop.memoryClockRate * 1e-6f, device_prop.memoryClockRate * 1e-3f);
			printf("\tMemory bus width : %d-bit\n",
					device_prop.memoryBusWidth);
			if (device_prop.l2CacheSize) {
				printf("\tL2 cache size: %.2f MB (%d bytes)\n",
						device_prop.l2CacheSize/std::pow(1024.0f, 2.0f), device_prop.l2CacheSize);
			}
			printf("\tMax texture dimension size (x, y, z):\n");
			printf(	"\t\t- 1D=(%d)\n"
					"\t\t- 2D=(%d, %d)\n"
					"\t\t- 3D=(%d, %d, %d)\n",
					device_prop.maxTexture1D,
					device_prop.maxTexture2D[0], device_prop.maxTexture2D[1],
					device_prop.maxTexture3D[0], device_prop.maxTexture3D[1], device_prop.maxTexture3D[2]);
			printf("\tMax layered texture size (dim) * layers:\n");
			printf(	"\t\t- 1D=(%d) * %d\n"
					"\t\t- 2D=(%d, %d) * %d\n",
					device_prop.maxTexture1DLayered[0], device_prop.maxTexture1DLayered[1],
					device_prop.maxTexture2DLayered[0], device_prop.maxTexture2DLayered[1], device_prop.maxTexture2DLayered[2]);
			printf("\tTotal amount of constant memory: %.2f KB (%lu bytes)\n",
					device_prop.totalConstMem/1024.0f, device_prop.totalConstMem);
			printf("\tTotal amount of shared memory per block: %.2f KB (%lu bytes)\n",
					device_prop.sharedMemPerBlock/1024.0f, device_prop.sharedMemPerBlock);
			printf("\tTotal number of registers available per block: %d\n",
					device_prop.regsPerBlock);
			printf("\tWarp size: %d\n",
					device_prop.warpSize);
			printf("\tSM (streaming multiprocessor) count: %d\n", device_prop.multiProcessorCount);
			printf("\tWarps per SM: %d\n", static_cast<int>(std::floor(device_prop.maxThreadsPerMultiProcessor/device_prop.warpSize)));
			printf("\tMaximum number of threads per SM: %d\n", device_prop.maxThreadsPerMultiProcessor);
			printf("\tMaximum number of threads per block: %d\n", device_prop.maxThreadsPerBlock);
			printf("\tMaximum number of threads total: %d\n", device_prop.multiProcessorCount * device_prop.maxThreadsPerMultiProcessor);
			printf("\tMaximum sizes of each dimension of a block: %d * %d * %d\n",
					device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1], device_prop.maxThreadsDim[2]);
			printf("\tMaximum sizes of each dimension of a grid: %d * %d * %d\n",
					device_prop.maxGridSize[0], device_prop.maxGridSize[1], device_prop.maxGridSize[2]);
			printf("\tMaximum memory pitch: %.2f GB (%lu bytes)\n",
					device_prop.memPitch/std::pow(1024.0f, 3.0f), device_prop.memPitch);
		}
	}
	printf("------------------------------------------------------------------------------\n");
}

__host__
void CheckCUDAErr(const std::initializer_list<const hipError_t>& errors) {
	#pragma unroll
	for (auto err: errors) {
		if (err != hipSuccess) {
			fprintf(stderr, "%s\n", hipGetErrorString(err));
			throw std::runtime_error {"CUDA error encountered."};
		}
	}
}

#endif
